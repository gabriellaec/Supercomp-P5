#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <thrust/transform.h>
#include <vector>
#include <string>
#include<bits/stdc++.h>
#include <omp.h>

using namespace std;

// ----- Pesos definidos ----- //
#define WMAT 2
#define WMIS -1
#define WGAP -1

// ----- Structs ----- //

struct combination {
    long value;
    vector<char> seq_a;
    vector<char> seq_b;
};

// ----- Functors ----- //

// Functor para o cálculo do score na GPU
struct meu_functor
{
    thrust::device_ptr<char> d_S;
    thrust::device_ptr<int> calc0;
    char letradeT;

    meu_functor( thrust::device_ptr<char> d_S_, char letradeT_, thrust::device_ptr<int>  calc0_) : d_S(d_S_), letradeT(letradeT_), calc0(calc0_) {};
    __host__ __device__
    int operator() (const int(&j) ){
        int diagonal, insercao;

        insercao = calc0[j] - 1;

        if (d_S[j] == letradeT) {
            diagonal = calc0[j-1] + WMAT;
        } else {
            diagonal = calc0[j-1] + WMIS;
        }

        int max = 0;
        if (diagonal > max) {
            max = diagonal;
        }

        if (insercao > max) {
            max = insercao;
        }

        return max;
    }
};

// ----- Funções ----- //

// Função que calcula os scores de todas as combinações do batch passado para a GPU 
// e retorna o melhor deles 
int gpu_score(vector<combination> &combinations){

    int maior = 0;

    for (auto& el : combinations){
        vector<char> S = el.seq_a;
        vector<char> T = el.seq_b;
        int N = el.seq_a.size();
        int M = el.seq_b.size();


        thrust::device_vector<int> calc[2]; // precisa ser um vector
        calc[0].resize(N+1);  // linha anterior
        calc[1].resize(N+1);  // resultado temporário da transformação

        // preenche a linha anterior com zeros
        thrust::fill(calc[0].begin(), calc[0].end(),0);

        // copia a seq S pra d_S
        thrust::device_vector<char> d_S(N);
        thrust::copy( S.begin(), S.begin()+N, d_S.begin());

        // counting iterators
        thrust::counting_iterator<int> c0(1);
        thrust::counting_iterator<int> c1(M+1);

        for (int j=0; j<M; j++)
        {
            char letradeT = T[j];
            thrust::transform(c0, c1, calc[1].begin()+1, meu_functor(d_S.data(), letradeT, calc[0].data()));  // calcula a diagonal e superior
            thrust::inclusive_scan(calc[1].begin()+1, calc[1].end(), calc[0].begin()+1, thrust::maximum<int>());  // calcula o lateral
        }

        // int max = thrust::reduce(calc[1].begin()+1, calc[1].end(),-1,thrust::maximum<int>());
        int max = calc[1].data()[calc[1].size()-1];

        if (max>maior) maior = max;
    }

    return maior;
}

// Fução usada para repartir o vetor em batches para passar para a GPU
void slicing(vector<combination>& arr, int X, int Y)
{
    auto start = arr.begin() + X;
    auto end = arr.begin() + Y + 1;
    vector<combination> result(Y - X + 1);
    copy(start, end, result.begin());
}


// Fução usada para gerar as subsequências
void gera_subseq(string seq, int start_point, int end_point, vector<vector<char>>& matriz_subseq){
    if (end_point > (int)seq.size())
      return;
    else if (start_point > end_point){
        gera_subseq(seq, 0, end_point+1, matriz_subseq);
    }else{
        if (start_point != end_point){
            vector<char> subseq;
            for (int j=start_point; j<end_point; j++){
                subseq.push_back(seq[j]);
            }
            matriz_subseq.push_back(subseq);
        }
        gera_subseq(seq, start_point+1, end_point, matriz_subseq);
    }
}


int main() {
    double init_time, final_time;
    init_time = omp_get_wtime();

// Lendo as sequências de um arquivo
    int N,M;
    string base;

    cin >> N >> M;
    string S_str;
    string T_str;

    cin >> S_str;
    cin >> T_str;
    
// Gerando as subsequências
    vector<vector<char>> subseqs_a;
    vector<vector<char>> subseqs_b;
    gera_subseq((S_str),0,1, subseqs_a);
    gera_subseq((T_str),0,1,subseqs_b);

    vector<char> T(T_str.begin(), T_str.end());
    vector<char> S(S_str.begin(), S_str.end());

// Calculando os scores
    vector<combination> combinations((long)subseqs_a.size()*(long)subseqs_b.size());;  

    long i=0;
    int melhor_valor = -1;
    int melhor_valor_g = -1;  // melhor valor global
    int val=-1;
    
    int max_possible = 0;
    int branch_bound_saves = 0;

    int max_vector_size = 600000;  // máximo definido para não estourar a memória
    int limit=max_vector_size/4;  // tamanho dos batches a serem enviados para a GPU


    for (auto& sub_a : subseqs_a){
        for (auto& sub_b : subseqs_b){
            if (sub_a.size() < sub_b.size())
                max_possible = sub_a.size()*2;
            else max_possible = sub_b.size()*2;


            if (max_possible > melhor_valor_g){
                combinations.push_back({i,sub_a, sub_b});
                i+=1;
            }else branch_bound_saves++;

            if (i>=max_vector_size){ // divisão em sub blocos para não estourar a memória
                
                #pragma omp parallel for reduction(max:melhor_valor)
                for (int index=0; index<combinations.size(); index+=limit){
                    slicing(combinations, index, index+limit-1); 
                    val = gpu_score(combinations);
                    if (val>melhor_valor)melhor_valor=val;
                } 

                if (melhor_valor > melhor_valor_g){
                    melhor_valor_g = melhor_valor;
                }
          
            i = 0;   
            combinations.clear();             
            }
        }
    }

    #pragma omp parallel for reduction(max:melhor_valor)
    for (int index=0; index<combinations.size(); index+=limit){ 
        if (index+limit>combinations.size()){  // para não acessar posições inválidas no vetor
            int start = index;  
            int end =combinations.size()-1;

            // cout << "start: " << start << endl;
            // cout << "end: " << end << endl;
            slicing(combinations, start, end);
            val = gpu_score(combinations);
        }else{
            slicing(combinations, index, index+limit-1);
            val = gpu_score(combinations);

            // cout << "start: " << index << endl;
            // cout << "end: " << index+limit << endl;        
        
        }
        // cout << "val: " << val << endl;
        if (val>melhor_valor)melhor_valor=val;
    }
    
        
    if (melhor_valor > melhor_valor_g){
        melhor_valor_g = melhor_valor;
    }

    cout << endl << "result: " << melhor_valor_g << endl;
    final_time = omp_get_wtime() - init_time;
    cout << "tempo: " << final_time << endl;
    cout << "branch_bound_saves: " << branch_bound_saves << endl;

    return 0;

}
