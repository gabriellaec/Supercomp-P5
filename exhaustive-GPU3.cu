#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <thrust/transform.h>
#include <vector>
#include <string>
#include<bits/stdc++.h>
#include <omp.h>

using namespace std;

// ----- Pesos definidos ----- //
#define WMAT 2
#define WMIS -1
#define WGAP -1

// ----- Structs ----- //

struct combination {
    long value;
    vector<char> seq_a;
    vector<char> seq_b;
};

// ----- Functors ----- //

// Functor para o cálculo do score na GPU
struct meu_functor
{
    thrust::device_ptr<char> d_S;
    thrust::device_ptr<int> calc0;
    char letradeT;

    meu_functor( thrust::device_ptr<char> d_S_, char letradeT_, thrust::device_ptr<int>  calc0_) : d_S(d_S_), letradeT(letradeT_), calc0(calc0_) {};
    __host__ __device__
    int operator() (const int(&j) ){
        int diagonal, insercao;

        insercao = calc0[j] - 1;

        if (d_S[j] == letradeT) {
            diagonal = calc0[j-1] + WMAT;
        } else {
            diagonal = calc0[j-1] + WMIS;
        }

        int max = 0;
        if (diagonal > max) {
            max = diagonal;
        }

        if (insercao > max) {
            max = insercao;
        }

        return max;
    }
};




// ----- Funções ----- //

// Função que calcula os scores de todas as combinações do batch passado para a GPU 
// e retorna o melhor deles 
int gpu_score(vector<vector<char>> subseqs_a, vector<vector<char>> subseqs_b, int global_max){

    int maior = 0;
    int max_possible=0;
     for (auto& S : subseqs_a){
      for (auto& T : subseqs_b){
        int N = S.size();
        int M = T.size();

        if (N>M) max_possible = M*2;
        else max_possible = N*2;
        if (maior<max_possible){


        thrust::device_vector<int> calc[2]; // precisa ser um vector
        calc[0].resize(N+1);  // linha anterior
        calc[1].resize(N+1);  // resultado temporário da transformação

        // preenche a linha anterior com zeros
        thrust::fill(calc[0].begin(), calc[0].end(),0);

        // copia a seq S pra d_S
        thrust::device_vector<char> d_S(N);
        thrust::copy( S.begin(), S.begin()+N, d_S.begin());

        // counting iterators
        thrust::counting_iterator<int> c0(1);
        thrust::counting_iterator<int> c1(M+1);

        for (int j=0; j<M; j++)
        {
            char letradeT = T[j];
            thrust::transform(c0, c1, calc[1].begin()+1, meu_functor(d_S.data(), letradeT, calc[0].data()));  // calcula a diagonal e superior
            thrust::inclusive_scan(calc[1].begin()+1, calc[1].end(), calc[0].begin()+1, thrust::maximum<int>());  // calcula o lateral
        }

        // int max = thrust::reduce(calc[1].begin()+1, calc[1].end(),-1,thrust::maximum<int>());
        int max = calc[1].data()[calc[1].size()-1];

        if (max>maior) maior = max;

        if (max >= global_max) return max;

        }

    }
    }

    return maior;
}

// Fução usada para gerar as subsequências
void gera_subseq(string seq, int start_point, int end_point, vector<vector<char>>& matriz_subseq){
    if (end_point > (int)seq.size())
      return;
    else if (start_point > end_point){
        gera_subseq(seq, 0, end_point+1, matriz_subseq);
    }else{
        if (start_point != end_point){
            vector<char> subseq;
            for (int j=start_point; j<end_point; j++){
                subseq.push_back(seq[j]);
            }
            matriz_subseq.push_back(subseq);
        }
        gera_subseq(seq, start_point+1, end_point, matriz_subseq);
    }
}

int main() {
    double init_time, final_time;
    init_time = omp_get_wtime();

// Lendo as sequências de um arquivo
    int N,M;
    string base;

    cin >> N >> M;
    
    string S_str;
    string T_str;

    cin >> S_str;
    cin >> T_str;

// Gerando as subsequências
    vector<vector<char>> subseqs_a;
    vector<vector<char>> subseqs_b;
    gera_subseq((S_str),0,1, subseqs_a);
    gera_subseq((T_str),0,1,subseqs_b);

    vector<char> T(T_str.begin(), T_str.end());
    vector<char> S(S_str.begin(), S_str.end());

// Calculando os scores
    int melhor_valor_g = -1;  // melhor valor global
    int global_max = 0;

    if (T.size() > S.size()) global_max = 2*S.size();
    else global_max = 2*T.size();

    melhor_valor_g = gpu_score(subseqs_a,subseqs_b, global_max);

    // cout << endl << "result: " << melhor_valor_g << endl;
    // final_time = omp_get_wtime() - init_time;
    // cout << "tempo: " << final_time << endl;

    cout << melhor_valor_g << endl;

    return 0;

}
