#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <thrust/transform.h>

using namespace std;

// ----- Pesos definidos ----- //
#define WMAT 2
#define WMIS -1
#define WGAP -1


struct meu_functor
{
    thrust::device_ptr<char> d_S;
    thrust::device_ptr<int> calc0;
    char letradeT;

    meu_functor( thrust::device_ptr<char> d_S_, char letradeT_, thrust::device_ptr<int>  calc0_) : d_S(d_S_), letradeT(letradeT_), calc0(calc0_) {};
    __host__ __device__
    int operator() (const int(&j) ){

        if (d_S[j] == letradeT)
            return calc0[j]+WMAT;
        else if (d_S[j] != letradeT)
            return calc0[j]+WMIS;  // mismatch
        else
            return calc0[j]+WGAP;
           
    }
};



int main() {

    int N,M;
    string base;

    cin >> N >> M;
    char *S;
    char *T;

    cin >> S;
    cin >> T;


    // int N,M;
    // char base;

    // cin >> N >> M;
    // char *S;
    // char *T;
    
    // // vector<char> S(N+1,0);
    // // vector<char> T(M,0);

    // cin >> S;
    // // for(int i = 0; i < N; i++){
    // //     S.push_back(base[i]);
    // // }
    // cin >> T;
    // for(int i = 0; i < M; i++){
    //     T.push_back(base[i]);
    // }

    // int N,M;
    // char *S = "AGCA";
    // char *T = "ACACA";
    // N = strlen(S);
    // M = strlen(T);

    thrust::device_vector<int> calc[2]; // precisa ser um vector
    calc[0].resize(N+1);  // linha anterior
    calc[1].resize(N+1);  // resultado temporário da transformação

    // preenche a linha anterior com zeros
    thrust::fill(calc[0].begin(), calc[0].end(),0);

    // copia a seq S pra d_S
    thrust::device_vector<char> d_S(N);
    thrust::copy(S, S+N, d_S.begin());

    thrust::counting_iterator<int> c0(1);
    thrust::counting_iterator<int> c1(M+1);

    // std::cout << d_S.data();

    for (int j=0; j<M; j++)
    {
        char letradeT = T[j];
        thrust::transform(c0, c1, calc[1].begin()+1, meu_functor(d_S.data(), letradeT, calc[0].data()));
        thrust::inclusive_scan(calc[1].begin()+1, calc[1].end(), calc[0].begin()+1, thrust::maximum<int>());
    }


    // for (int i=0; i< N; i++)
    //      cout << "maximo: " << el << endl;

    int max = thrust::reduce(calc[1].begin(), calc[1].end(), thrust::maximum<int>());
    cout << "maximo: " << max;

    return 0;

}


// Tratar tamanhos diferentes no functor
// c1 e c2 são counting iterators pra resolver o problema de que precisa ter 2 vectors de tamanhos =s


// functor pode receber device_ptr como parametro para apontar o inicio do device_vector

// diagonal e superior vem de calc[0]
// comparar com letradeT pra ver se é match ou mismatch

// na linha do functor
// posicao de S chamada de j
// para usar declarar: operator() int(&j) const


// nvcc -arch=sm_70 -std=c++14 p5.cu -o p5