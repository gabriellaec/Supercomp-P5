#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <cmath>

int main() {
    char *S = "GAA"
    char *T = "GAA"
    N = strlen(S)
    M = strlen(T)

    device_vector<int> calc[2];
    calc[0].resize(N+1);  // linha anterior
    calc[1].resize(N+1);  // resultado temporário da transformação

    // preenche a linha anterior com zeros
    thrust::fill(calc[0].begin(), calc[0].end(),0);

    // copia a seq S pra d_S
    device_vector<char> d_S();
    thrust::copy(S, S+N, d_S.begin());

    for (int j=0; j<M; j++)
    {
        char letradeT = T[j];
        thrust::transform(c0, c1, calc[1].begin()+1, meu_functor(d_S.data(), letradeT, calc[0].data()));
        thrust::inclusive_scan(calc[1].begin+1, calc[1].end(), calc[0].begin()+1, thrust::maximum<int>());
    }

}


// Tratar tamanhos diferentes no functor
// c1 e c2 são counting iterators pra resolver o problema de que precisa ter 2 vectors de tamanhos =s


// functor pode receber device_ptr como parametro para apontar o inicio do device_vector

// diagonal e superior vem de calc[0]
// comparar com letradeT pra ver se é match ou mismatch

// na linha do functor
// posicao de S chamada de j
// para usar declarar: operator() int(&j) const