#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <thrust/transform.h>
#include <vector>
#include <string>
#include<bits/stdc++.h>
#include <omp.h>

using namespace std;

// ----- Pesos definidos ----- //
#define WMAT 2
#define WMIS -1
#define WGAP -1

// ----- Structs ----- //
struct item {
    long item_score;
    vector<char> seq_a;
    vector<char> seq_b;
};

struct combination {
    long value;
    vector<char> seq_a;
    vector<char> seq_b;
};



// ----- Functors ----- //
struct meu_functor
{
    thrust::device_ptr<char> d_S;
    thrust::device_ptr<int> calc0;
    char letradeT;

    meu_functor( thrust::device_ptr<char> d_S_, char letradeT_, thrust::device_ptr<int>  calc0_) : d_S(d_S_), letradeT(letradeT_), calc0(calc0_) {};
    __host__ __device__
    int operator() (const int(&j) ){

        if (d_S[j] == letradeT)
            return calc0[j]+WMAT;
        else if (d_S[j] != letradeT)
            return calc0[j]+WMIS;  // mismatch
        else
            return calc0[j]+WGAP;
           
    }
};


int gpu_score(vector<char> S, vector<char> T, int N, int M){
    thrust::device_vector<int> calc[2]; // precisa ser um vector
    calc[0].resize(N+1);  // linha anterior
    calc[1].resize(N+1);  // resultado temporário da transformação

    // preenche a linha anterior com zeros
    thrust::fill(calc[0].begin(), calc[0].end(),0);

    // copia a seq S pra d_S
    thrust::device_vector<char> d_S(N);
    thrust::copy( S.begin(), S.begin()+N, d_S.begin());

    thrust::counting_iterator<int> c0(1);
    thrust::counting_iterator<int> c1(M+1);

    // std::cout << d_S.data();

    for (int j=0; j<M; j++)
    {
        char letradeT = T[j];
        thrust::transform(c0, c1, calc[1].begin()+1, meu_functor(d_S.data(), letradeT, calc[0].data()));
        thrust::inclusive_scan(calc[1].begin()+1, calc[1].end(), calc[0].begin()+1, thrust::maximum<int>());
    }


    // for (int i=0; i< N; i++)
    //      cout << "maximo: " << el << endl;

    int max = calc[1].data()[calc[1].size()-1];

    return max;
}



// void gera_subseq(string seq, long start_point, long end_point, vector<char>& subseqs){
//     if (end_point > (long)seq.size())
//       return;
//     else if (start_point > end_point){
//         gera_subseq(seq, 0, end_point+1, subseqs);
//     }else{
//         if (start_point != end_point){
//             string subseq;
//             for (long j=start_point; j<end_point; j++){
//                 subseq.push_back(seq[j]);
//             }
//             subseqs.push_back(subseq);
//         }
//         gera_subseq(seq, start_point+1, end_point, subseqs);
//     }
// }


void gera_subseq(string input, vector<vector<char>> &all){
    for (int j=0; j<(int)input.size(); j++){
        for(int i=0; i<(int)input.size(); i++){
            string sub = input.substr(i,j);
            vector<char> sub_char(sub.begin(), sub.end());
            if (!(sub_char.size()) < 1)
            all.push_back(sub_char);
        }
    }
}



int main() {
    int N,M;
    string base;

    cin >> N >> M;
    string S_str;
    string T_str;

    cin >> S_str;
    cin >> T_str;
    
    // char *S;
    // char *T;

    // cin >> base;
    // for(int i = 0; i < N; i++){
    //     S.push_back(base[i]);
    // }
    // cin >> base;
    // for(int i = 0; i < M; i++){
    //     T.push_back(base[i]);
    // }

    // int N,M;
    // char *S = "AGCA";
    // char *T = "ACACA";
    // N = strlen(S);
    // M = strlen(T);


    vector<vector<char>> subseqs_a;
    vector<vector<char>> subseqs_b;
    gera_subseq(S_str, subseqs_a);
    gera_subseq(T_str,subseqs_b);


    
    // strcpy(T, T_str);

    vector<char> T(T_str.begin(), T_str.end());
    vector<char> S(S_str.begin(), S_str.end());

///////////////////////////////////////

    item melhor, sw_atual;    
    vector<combination> combinations((long)subseqs_a.size()*(long)subseqs_b.size());;  


    // cout << (long)subseqs_a.size()*(long)subseqs_b.size() << " combinations" << endl;
 
    long i=0;
    int melhor_valor = -1;
    int melhor_valor_g = -1;
    int val;

    for (auto& sub_a : subseqs_a){
        for (auto& sub_b : subseqs_b){
            combinations.push_back({i,sub_a, sub_b});
            i+=1;
            if (i>=600000){ // divisão em sub blocos para não estourar o vetor
                #pragma omp parallel for reduction(max:melhor_valor)
                for (auto& el : combinations){ 
                    val = gpu_score(el.seq_a, el.seq_b,(el.seq_a).size(),(el.seq_b).size());
                    if (val>melhor_valor)melhor_valor=val;
                } 

                for (int i=0; i<(int)resultados.size(); i++){
                    if (melhor_valor > melhor_valor_g){
                        melhor_valor_g = melhor_valor;
                    }
                }
            // -----------------------------------//
            i = 0;   
            combinations.clear();             
            }
        }
    }


    #pragma omp parallel for reduction(max:melhor_valor)
    for (auto& el : combinations){ 
            val = gpu_score(el.seq_a, el.seq_b,(el.seq_a).size(),(el.seq_b).size());
            if (val>melhor_valor)melhor_valor=val;
            
    }
    
        
    for (int i=0; i<(int)resultados.size(); i++){
        if (melhor_valor > melhor_valor_g){
            melhor_valor_g = melhor_valor;
        }
    }

//////////////////////////////////////
    // vector<int> resultados((long)combinations.size());

    // cout << "--------------------";
    // #pragma omp parallel for shared(resultados) 
    // for (auto& el : combinations){
    //     resultados[el.value] = gpu_score(el.seq_a, el.seq_b,(el.seq_a).size(),(el.seq_b).size());
    // }
//////////////////////////////////////


    // long melhor_valor=-1;
    // for (long i=0; i<(long)resultados.size(); i++){
    //     if (resultados[i] > melhor_valor){
    //         melhor_valor = resultados[i];
    //     }
    // }



    cout << "maximo: " << melhor_valor_g << endl;

    return 0;

}


// Tratar tamanhos diferentes no functor
// c1 e c2 são counting iterators pra resolver o problema de que precisa ter 2 vectors de tamanhos =s


// functor pode receber device_ptr como parametro para apontar o inicio do device_vector

// diagonal e superior vem de calc[0]
// comparar com letradeT pra ver se é match ou mismatch

// na linha do functor
// posicao de S chamada de j
// para usar declarar: operator() int(&j) const


// nvcc -arch=sm_70 -std=c++14 p5.cu -o p5